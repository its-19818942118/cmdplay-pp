#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "Version.hpp"
#include "audio/AudioEngine.hpp"
#include "audio/AudioException.hpp"
#include "Instance.hpp"
#include "VideoPlayer.hpp"
#include "ConsoleUtils.hpp"
#include <string>

constexpr const char* BRIGHTNESS_LEVELS = " .-+*wGHM#&%";


int main(int argc, char* argv[])
{
	std::cout << "cmdplay++ version " << cmdplay::VERSION << std::endl;
	try
	{
		cmdplay::Instance::AudioEngine = new cmdplay::audio::AudioEngine(-1, 44100, 0);
	}
	catch (cmdplay::audio::AudioException& ex)
	{
		std::cout << "Failed to initialize audio! Error message: " << ex.GetExMessage() << std::endl;
	}
	std::string filenameInput;
	if (argc > 1)
	{
		filenameInput = std::string(argv[1]);
	}
	else
	{
		std::cout << "Enter filename: ";
		std::getline(std::cin, filenameInput);
		if (filenameInput.length() == 0)
		{
			std::cout << "Invalid filename" << std::endl;
			return EXIT_FAILURE;
		}
	}

	if (filenameInput.length() > 2) // Remove quotation marks if present
	{
		if (filenameInput[0] == '"')
			filenameInput = filenameInput.substr(1);
		if (filenameInput[filenameInput.length() - 1] == '"')
			filenameInput = filenameInput.substr(0, filenameInput.length() - 1);
	}

	cmdplay::VideoPlayer player(filenameInput, BRIGHTNESS_LEVELS);
	player.LoadVideo();
	player.Enter();

	return EXIT_SUCCESS;
}