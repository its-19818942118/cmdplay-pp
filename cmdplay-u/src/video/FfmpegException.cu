#include "FfmpegException.cuh"

cmdplay::video::FfmpegException::FfmpegException(const std::string& msg)
{
	m_msg = msg;
}

std::string& cmdplay::video::FfmpegException::GetMessage()
{
	return m_msg;
}
