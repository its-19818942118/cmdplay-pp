#include "hip/hip_runtime.h"
#include "gpuAsciifier.cuh"
#include <math.h>


cmdplay::gpuAsciiFier::gpuAsciiFier(const std::string& brightnessLevels, int frameWidth, int frameHeight,
	bool useColors, bool useColorDithering, bool useTextDithering, bool useAccurateColors, bool useAccurateColorsFullPixel):
	m_brightnessLevels(brightnessLevels), m_frameWidth(frameWidth), m_frameHeight(frameHeight),
	m_useColorDithering(useColorDithering), m_useTextDithering(useTextDithering),
	m_brightnessLevelCount(static_cast<uint8_t>(brightnessLevels.length())), m_useColors(useColors),
	m_useAccurateColors(useAccurateColors), m_useAccurateColorsFullPixel(useAccurateColorsFullPixel)
{
	m_framepixelbytescount = m_frameWidth * m_frameHeight * 4;
	m_framebuffersize = (m_frameWidth+1) * m_frameHeight;
	m_frameWidthWithStride = m_frameWidth;
}

std::string cmdplay::gpuAsciiFier::BuildFrame(uint8_t * rgbData) {
	char* framechars;
	uint8_t* rgb;
	char* brightnesslevel;
	char* d_framechars;

	int rgbsize, framecharssize, brightnesslevelsize;
	rgbsize = sizeof(unsigned char) * m_framepixelbytescount;
	framecharssize = sizeof(char) * m_framebuffersize;
	brightnesslevelsize = sizeof(char) * m_brightnessLevelCount;

	uint8_t* d_rgb;
	char * d_brightnessLevels;

	

	hipHostMalloc((void**)&framechars, framecharssize, hipHostMallocDefault);
	hipHostMalloc((void**)&rgb, rgbsize, hipHostMallocDefault);
	hipHostMalloc((void**)&brightnesslevel,brightnesslevelsize, hipHostMallocDefault);

	hipMalloc((void**)&d_framechars, framecharssize);
	hipMalloc((void**)&d_rgb, rgbsize);
	hipMalloc(&d_brightnessLevels, brightnesslevelsize);

	rgb = rgbData;

	brightnesslevel = (char*)m_brightnessLevels.c_str();

	for (int i = 1; i < m_framebuffersize / (m_frameWidthWithStride + 1) + 1; ++i) {
		framechars[i * (m_frameWidthWithStride + 1) - 1] = '\n';

	}
	framechars[m_framebuffersize] = '\0';

	hipMemcpy(d_framechars, framechars, framecharssize, hipMemcpyHostToDevice);
	hipMemcpy(d_rgb, rgb, rgbsize, hipMemcpyHostToDevice);
	hipMemcpy(d_brightnessLevels, brightnesslevel, brightnesslevelsize, hipMemcpyHostToDevice);

	asciifier<< <m_frameHeight*m_frameWidth/256 + 1, 256>> > (d_rgb, d_framechars,d_brightnessLevels, m_brightnessLevelCount, m_frameWidth);

	hipMemcpy(framechars, d_framechars, framecharssize, hipMemcpyHostToDevice);

	hipFree(d_framechars);
	hipFree(d_rgb);
	hipFree(d_brightnessLevels);

	hipHostFree(framechars);
	hipHostFree(rgb);
	hipHostFree(brightnesslevel);

	return std::string(framechars);
}

__global__ void asciifier(uint8_t * rgbData, char * framechars, char * brightnesslevel, int brightnesslevelcount, int width) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int byteindex = index * 4;
	float check = static_cast<float>(index) / width;
	int frameindex = index + (int)floor(check);

	float r, g, b;
	r = (float)(int)rgbData[byteindex] / 255;
	g = (float)(int)rgbData[byteindex + 1] / 255;
	b = (float)(int)rgbData[byteindex + 2] / 255;

	int brightnessindex = (0.299 * r + 0.587 * g + 0.114 * b) * brightnesslevelcount;

	if (brightnessindex < 0) {
		brightnessindex = 0;
	}
	else if (brightnessindex >= brightnesslevelcount) {
		brightnessindex = brightnesslevelcount - 1;
	}

	framechars[frameindex] = brightnesslevel[brightnessindex];
}